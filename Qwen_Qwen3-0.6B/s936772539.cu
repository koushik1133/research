#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void odd_even_transpose(int *a, int *b, int N) {
  int i,j;
  __syncthreads();
  for (i=0; i < N; i++) {
    for (j=0; j < N; j++) {
      if (i == j) {
        b[i] = a[j];
      } else {
        b[i] = a[j];
      }
    }
  }
}
__global__ void bubble_sort(int *a, int *b, int N) {
  int i,j;
  __syncthreads();
  for (i=0; i < N; i++) {
    for (j=0; j < N; j++) {
      if (a[i] > a[j]) {
        b[i] = a[j];
        a[j] = a[i];
      }
    }
  }
}
__global__ void transpose(int *a, int *b, int N) {
  int i,j;
  __syncthreads();
  for (i=0; i < N; i++) {
    for (j=0; j < N; j++) {
      if (i == j) {
        b[i] = a[j];
      } else {
        b[i] = a[j];
      }
    }
  }
}
__global__ void swap(int *a, int *b, int i, int j) {
  int k;
  __syncthreads();
  for (k=0; k < N; k++) {
    if (k == i) {
      a[k] = b[k];
    } else if (k == j) {
      b[k] = a[k];
    }
  }
}
__global__ void swap2(int *a, int *b, int i, int j) {
  int k;
  __syncthreads();
  for (k=0; k < N; k++) {
    if (k == i) {
      a[k] = b[k];
    } else if (k == j) {
      b[k] = a[k];
    }
  }
}
__global__ void swap3(int *a, int *b, int i, int j) {
  int k;
  __syncthreads();
  for (k=0; k < N; k++) {
    if (k == i) {
      a[k] = b[k];
    } else if (k == j) {
      b[k] = a[k];
    }
  }
}

int main() {
  int a,b;
  for(int j=1;j<=9;j++){
    for(int i=1;i<=9;i++)
      scanf("%d",&a);
  }
  hipMalloc(&a, N * sizeof(int));
  hipMalloc(&b, N * sizeof(int));
  hipMemcpy(a, input, N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(b, input, N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(a, input, N * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(b, input, N * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(a, input, N * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(b, input, N * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(a, input, N * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(b, input, N * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(a, input, N * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(b, input, N * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(a, input, N * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(b, input, N * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(a, input, N * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(b, input, N * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(a, input, N * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(b, input, N * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(a, input, N * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(b, input, N * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(a, input, N * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(b, input, N * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(a, input, N * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(b, input, N * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(a, input, N * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(b, input, N * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(a, input, N * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(b, input, N * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(a, input, N * sizeof