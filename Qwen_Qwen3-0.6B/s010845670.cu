#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void odd_even_sort(int *h, int N){
  int i = threadIdx.x;
  int j = threadIdx.y;
  int k = threadIdx.z;
  int offset = (i + j + k) % 2;
  if (offset == 0) {
    for (int m = 0; m < N; m += 2) {
      h[m] = h[m + 1];
    }
  } else {
    for (int m = 1; m < N; m += 2) {
      h[m] = h[m - 1];
    }
  }
}

int main() {
  int N = 10;
  int *h = (int*)hipMalloc(&N, sizeof(int));
  hipMemcpy(h, h, N, hipMemcpyHostToDevice);
  for (int i = 0; i < N; i++) {
    scanf("%d", &h[i]);
  }
  hipMemcpy(h, h, N, hipMemcpyDeviceToHost);
  hipFree(h);
  return 0;
}

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void odd_even_sort(int *h, int N) {
  int i = threadIdx.x;
  int j = threadIdx.y;
  int k = threadIdx.z;
  int offset = (i + j + k) % 2;
  if (offset == 0) {
    for (int m = 0; m < N; m += 2) {
      h[m] = h[m + 1];
    }
  } else {
    for (int m = 1; m < N; m += 2) {
      h[m] = h[m - 1];
    }
  }
}

int main() {
  int N = 10;
  int *h = (int*)hipMalloc(&N, sizeof(int));
  hipMemcpy(h, h, N, hipMemcpyHostToDevice);
  for (int i = 0; i < N; i++) {
    scanf("%d", &h[i]);
  }
  hipMemcpy(h, h, N, hipMemcpyDeviceToHost);
  hipFree(h);
  return 0;
}

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void odd_even_sort(int *h, int N) {
  int i = threadIdx.x;
  int j = threadIdx.y;
  int k = threadIdx.z;
  int offset = (i + j + k) % 2;
  if (offset == 0) {
    for (int m = 0; m < N; m += 2) {
      h[m] = h[m + 1];
    }
  } else {
    for (int m = 1; m < N; m += 2) {
      h[m] = h[m - 1];
    }
  }
}

int main() {
  int N = 10;
  int *h = (int*)hipMalloc(&N, sizeof(int));
  hipMemcpy(h, h, N, hipMemcpyHostToDevice);
  for (int i = 0; i < N; i++) {
    scanf("%d", &h[i]);
  }
  hipMemcpy(h, h, N, hipMemcpyDeviceToHost);
  hipFree(h);
  return 0;
}

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void odd_even_sort(int *h, int N) {
  int i = threadIdx.x;
  int j = threadIdx.y;
  int k = threadIdx.z;
  int offset = (i + j + k) % 2;
  if (offset == 0) {
    for (int m = 0; m < N; m += 2) {
      h[m] = h[m + 1];
    }
  } else {
    for (int m = 1; m < N; m += 2) {
      h[m] = h[m - 1];
    }
  }
}

int main() {
  int N = 10;
  int *h = (int*)hipMalloc(&N, sizeof(int));
  hipMemcpy(h, h, N, hipMemcpyHostToDevice);
  for (int i = 0; i < N; i++) {
    scanf("%d", &h[i]);
  }
  hipMemcpy(h, h, N, hipMemcpyDeviceToHost);
  hipFree(h);
  return 0;
}

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void odd_even_sort(int *h, int N) {
  int i = threadIdx.x;
  int j = threadIdx.y;
  int k = threadIdx.z;
  int offset = (i + j + k) % 2;
  if (offset == 0) {
    for (int m = 0; m < N; m += 2) {
      h