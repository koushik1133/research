#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void odd_even_sort(int *arr, int size) {
    int i = threadIdx.x;
    int j = threadIdx.y;
    int k = threadIdx.z;

    if (i == 0) {
        if (j == 0) {
            if (k == 0) {
                arr[i] = arr[i] + 1;
            } else {
                arr[i] = arr[i] - 1;
            }
        } else {
            if (k == 0) {
                arr[i] = arr[i] + 1;
            } else {
                arr[i] = arr[i] - 1;
            }
        }
    } else {
        if (j == 0) {
            if (k == 0) {
                arr[i] = arr[i] + 1;
            } else {
                arr[i] = arr[i] - 1;
            }
        } else {
            if (k == 0) {
                arr[i] = arr[i] + 1;
            } else {
                arr[i] = arr[i] - 1;
            }
        }
    }
}

int main() {
    int size = 10;
    int *arr = (int *)malloc(size * sizeof(int));
    hipMalloc(&arr, size * sizeof(int));
    hipMemcpy(arr, input_array, size * sizeof(int), hipMemcpyHostToDevice);

    for (int i = 0; i < size; ++i) {
        arr[i] = input_array[i];
    }

    __launch_bounds__(1, size);
    odd_even_sort(arr, size);
    hipMemcpy(arr, output_array, size * sizeof(int), hipMemcpyDeviceToHost);

    free(arr);
    return 0;
}

#include <iostream>
#include <queue>

using namespace std;

int main() {
    priority_queue<int> pq;

    for (int i = 0; i < 10; ++i) {
        int height;
        cin >> height;
        pq.emplace(height);
    }

    for (int i = 0; i <= 2; ++i) {
        std::cout << pq.top() << endl;
        pq.pop();
    }

    return 0;
}

CUDA code:

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void odd_even_sort(int *arr, int size) {
    int i = threadIdx.x;
    int j = threadIdx.y;
    int k = threadIdx.z;

    if (i == 0) {
        if (j == 0) {
            if (k == 0) {
                arr[i] = arr[i] + 1;
            } else {
                arr[i] = arr[i] - 1;
            }
        } else {
            if (k == 0) {
                arr[i] = arr[i] + 1;
            } else {
                arr[i] = arr[i] - 1;
            }
        }
    } else {
        if (j == 0) {
            if (k == 0) {
                arr[i] = arr[i] + 1;
            } else {
                arr[i] = arr[i] - 1;
            }
        } else {
            if (k == 0) {
                arr[i] = arr[i] + 1;
            } else {
                arr[i] = arr[i] - 1;
            }
        }
    }
}

int main() {
    int size = 10;
    int *arr = (int *)malloc(size * sizeof(int));
    hipMalloc(&arr, size * sizeof(int));
    hipMemcpy(arr, input_array, size * sizeof(int), hipMemcpyHostToDevice);

    for (int i = 0; i < size; ++i) {
        arr[i] = input_array[i];
    }

    __launch_bounds__(1, size);
    odd_even_sort(arr, size);
    hipMemcpy(arr, output_array, size * sizeof(int), hipMemcpyDeviceToHost);

    free(arr);
    return 0;
}

#include <iostream>
#include <queue>

using namespace std;

int main() {
    priority_queue<int> pq;

    for (int i = 0; i < 10; ++i) {
        int height;
        cin >> height;
        pq.emplace(height);
    }

    for (int i = 0; i <= 2; ++i) {
        std::cout << pq.top() << endl;
        pq.pop();
    }

    return 0;
}

CUDA code:

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void odd_even_sort(int *arr, int size) {
    int i = threadIdx.x;
    int j = threadIdx.y;
    int k = threadIdx.z;

    if (i == 0) {
        if (j == 0) {
            if (k ==