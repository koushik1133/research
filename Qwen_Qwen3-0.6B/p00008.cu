#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void odd_even_sort(int *A, int N) {
    int i, j, k;
    __syncthreads();
    if (i < N / 2) {
        for (j = 0; j < N; j++) {
            A[i][j] = A[i][j] + A[i][j + 1];
        }
    } else {
        for (j = 0; j < N; j++) {
            A[i][j] = A[i][j] - A[i][j - 1];
        }
    }
}

int main() {
    int A[N][N], B[N][N], C[N][N];
    hipMalloc(&A, N * N * sizeof(int));
    hipMalloc(&B, N * N * sizeof(int));
    hipMalloc(&C, N * N * sizeof(int));
    hipMemcpy(A, A, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(B, B, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(C, C, N * N * sizeof(int), hipMemcpyHostToDevice);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            C[i][j] = 0;
            for (int k = 0; k < N; k++) {
                C[i][j] += A[i][k] * B[k][j];
            }
        }
    }

    printf("C[0][0] = %d\n", C[0][0]);
    hipFree(A);
    hipFree(B);
    hipFree(C);
    return 0;
}

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void odd_even_sort(int *A, int N) {
    int i, j, k;
    __syncthreads();
    if (i < N / 2) {
        for (j = 0; j < N; j++) {
            A[i][j] = A[i][j] + A[i][j + 1];
        }
    } else {
        for (j = 0; j < N; j++) {
            A[i][j] = A[i][j] - A[i][j - 1];
        }
    }
}

int main() {
    int A[N][N], B[N][N], C[N][N];
    hipMalloc(&A, N * N * sizeof(int));
    hipMalloc(&B, N * N * sizeof(int));
    hipMalloc(&C, N * N * sizeof(int));
    hipMemcpy(A, A, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(B, B, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(C, C, N * N * sizeof(int), hipMemcpyHostToDevice);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            C[i][j] = 0;
            for (int k = 0; k < N; k++) {
                C[i][j] += A[i][k] * B[k][j];
            }
        }
    }

    printf("C[0][0] = %d\n", C[0][0]);
    hipFree(A);
    hipFree(B);
    hipFree(C);
    return 0;
}

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void odd_even_sort(int *A, int N) {
    int i, j, k;
    __syncthreads();
    if (i < N / 2) {
        for (j = 0; j < N; j++) {
            A[i][j] = A[i][j] + A[i][j + 1];
        }
    } else {
        for (j = 0; j < N; j++) {
            A[i][j] = A[i][j] - A[i][j - 1];
        }
    }
}

int main() {
    int A[N][N], B[N][N], C[N][N];
    hipMalloc(&A, N * N * sizeof(int));
    hipMalloc(&B, N * N * sizeof(int));
    hipMalloc(&C, N * N * sizeof(int));
    hipMemcpy(A, A, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(B, B, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(C, C, N * N * sizeof(int), hipMemcpyHostToDevice);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            C[i][j] = 0;