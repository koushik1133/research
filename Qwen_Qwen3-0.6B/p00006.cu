#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vector_add(float *a, float *b, float *result, int n) {
    int i = threadIdx.x;
    if (i < n) {
        result[i] = a[i] + b[i];
    }
}

int main() {
    float a[N] = {1.0, 2.0, 3.0, 4.0, 5.0};
    float b[N] = {10.0, 20.0, 30.0, 40.0, 50.0};
    float result[N];

    hipMalloc(&a, N * sizeof(float));
    hipMalloc(&b, N * sizeof(float));
    hipMalloc(&result, N * sizeof(float));

    hipMemcpy(a, a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b, b, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(result, result, N * sizeof(float), hipMemcpyHostToDevice);

    printf("Result vector:\n");
    for (int i = 0; i < N; i++) {
        printf("%.2f ", result[i]);
    }
    printf("\n");

    hipFree(a);
    hipFree(b);
    hipFree(result);

    return 0;
}

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vector_add(float *a, float *b, float *result, int n) {
    int i = threadIdx.x;
    if (i < n) {
        result[i] = a[i] + b[i];
    }
}

int main() {
    float a[N] = {1.0, 2.0, 3.0, 4.0, 5.0};
    float b[N] = {10.0, 20.0, 30.0, 40.0, 50.0};
    float result[N];

    hipMalloc(&a, N * sizeof(float));
    hipMalloc(&b, N * sizeof(float));
    hipMalloc(&result, N * sizeof(float));

    hipMemcpy(a, a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b, b, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(result, result, N * sizeof(float), hipMemcpyHostToDevice);

    printf("Result vector:\n");
    for (int i = 0; i < N; i++) {
        printf("%.2f ", result[i]);
    }
    printf("\n");

    hipFree(a);
    hipFree(b);
    hipFree(result);

    return 0;
}

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vector_add(float *a, float *b, float *result, int n) {
    int i = threadIdx.x;
    if (i < n) {
        result[i] = a[i] + b[i];
    }
}

int main() {
    float a[N] = {1.0, 2.0, 3.0, 4.0, 5.0};
    float b[N] = {10.0, 20.0, 30.0, 40.0, 50.0};
    float result[N];

    hipMalloc(&a, N * sizeof(float));
    hipMalloc(&b, N * sizeof(float));
    hipMalloc(&result, N * sizeof(float));

    hipMemcpy(a, a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b, b, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(result, result, N * sizeof(float), hipMemcpyHostToDevice);

    printf("Result vector:\n");
    for (int i = 0; i < N; i++) {
        printf("%.2f ", result[i]);
    }
    printf("\n");

    hipFree(a);
    hipFree(b);
    hipFree(result);

    return 0;
}

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vector_add(float *a, float *b, float *result, int n) {
    int i = threadIdx.x;
    if (i < n) {
        result[i] = a[i] + b[i];
    }
}

int main() {
    float a[N] = {1.0, 2.0, 3.0, 4.0, 5.0};
    float b[N] = {10.0, 20.0, 30.0, 40.0, 50.0};
    float result[N];

    hipMalloc(&a, N * sizeof(float));
    hipMalloc(&b, N * sizeof(float));
    hipMalloc(&result, N * sizeof(float));

    hipMemcpy(a, a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy