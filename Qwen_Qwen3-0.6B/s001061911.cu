#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void odd_even_sort(int *arr, int len) {
    int i = threadIdx.x;
    int j = threadIdx.y;
    int k = threadIdx.z;

    if (i < len && j < len && k < len) {
        if (arr[i] < arr[j])
            swap(arr[i], arr[j]);
    }
}

int main() {
    int len = 10;
    int arr[len];
    hipMalloc(&arr, len * sizeof(int));
    hipMemcpy(arr, arr, len * sizeof(int), hipMemcpyHostToDevice);
    for (int i = 0; i < len; i++)
        for (int j = 0; j < len; j++)
            if (arr[i] < arr[j])
                swap(arr[i], arr[j]);
    hipMemcpy(arr, arr, len * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(arr);
    return 0;
}

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void odd_even_sort(int *arr, int len) {
    int i = threadIdx.x;
    int j = threadIdx.y;
    int k = threadIdx.z;

    if (i < len && j < len && k < len) {
        if (arr[i] < arr[j])
            swap(arr[i], arr[j]);
    }
}

int main() {
    int len = 10;
    int arr[len];
    hipMalloc(&arr, len * sizeof(int));
    hipMemcpy(arr, arr, len * sizeof(int), hipMemcpyHostToDevice);
    for (int i = 0; i < len; i++)
        for (int j = 0; j < len; j++)
            if (arr[i] < arr[j])
                swap(arr[i], arr[j]);
    hipMemcpy(arr, arr, len * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(arr);
    return 0;
}

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void odd_even_sort(int *arr, int len) {
    int i = threadIdx.x;
    int j = threadIdx.y;
    int k = threadIdx.z;

    if (i < len && j < len && k < len) {
        if (arr[i] < arr[j])
            swap(arr[i], arr[j]);
    }
}

int main() {
    int len = 10;
    int arr[len];
    hipMalloc(&arr, len * sizeof(int));
    hipMemcpy(arr, arr, len * sizeof(int), hipMemcpyHostToDevice);
    for (int i = 0; i < len; i++)
        for (int j = 0; j < len; j++)
            if (arr[i] < arr[j])
                swap(arr[i], arr[j]);
    hipMemcpy(arr, arr, len * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(arr);
    return 0;
}

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void odd_even_sort(int *arr, int len) {
    int i = threadIdx.x;
    int j = threadIdx.y;
    int k = threadIdx.z;

    if (i < len && j < len && k < len) {
        if (arr[i] < arr[j])
            swap(arr[i], arr[j]);
    }
}

int main() {
    int len = 10;
    int arr[len];
    hipMalloc(&arr, len * sizeof(int));
    hipMemcpy(arr, arr, len * sizeof(int), hipMemcpyHostToDevice);
    for (int i = 0; i < len; i++)
        for (int j = 0; j < len; j++)
            if (arr[i] < arr[j])
                swap(arr[i], arr[j]);
    hipMemcpy(arr, arr, len * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(arr);
    return 0;
}

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void odd_even_sort(int *arr, int len) {
    int i = threadIdx.x;
    int j = threadIdx.y;
    int k = threadIdx.z;

    if (i < len && j < len && k < len) {
        if (arr[i] < arr[j])
            swap(arr[i], arr[j]);
    }
}

int main() {
    int len = 10;
    int arr[len];
    hipMalloc(&arr, len * sizeof(int));
    hipMemcpy(arr, arr, len * sizeof(int), hipMemcpyHostToDevice);
    for (int i = 0; i < len; i++)
        for (int j = 0; j < len; j++)
            if (arr[i] < arr[j])
                swap(arr[i], arr[j]);
    hipMemcpy(arr, arr, len * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(arr);
    return 0;
}

#include <hip/hip_runtime.h>
#include <stdio.h>

__global