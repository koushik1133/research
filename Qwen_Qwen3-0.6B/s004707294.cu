#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
 
__global__ void odd_even_sort(int *a, int N)
{
   int idx = threadIdx.x;
   int offset = idx * 2;
   int i = offset;
   int j = offset + 1;
   int k = offset + 2;
   int temp;
   if (idx == 0) {
       temp = a[i];
       a[i] = a[j];
       a[j] = a[k];
       a[k] = temp;
   } else {
       if (idx == 1) {
           temp = a[i];
           a[i] = a[j];
           a[j] = a[k];
           a[k] = temp;
       } else {
           if (idx == 2) {
               temp = a[i];
               a[i] = a[j];
               a[j] = a[k];
               a[k] = temp;
           }
       }
   }
}
 
int main() {
   int a[10], i;
   hipMalloc(&a, 10 * sizeof(int));
   hipMemcpy(a, a, 10 * sizeof(int), hipMemcpyHostToDevice);
   for (i = 0; i < 10; i++) {
       scanf("%d", &a[i]);
   }
   hipFree(a);
   return 0;
}

#include <hip/hip_runtime.h>
#include <stdio.h>
 
__global__ void odd_even_sort(int *a, int N)
{
   int idx = threadIdx.x;
   int offset = idx * 2;
   int i = offset;
   int j = offset + 1;
   int k = offset + 2;
   int temp;
   if (idx == 0) {
       temp = a[i];
       a[i] = a[j];
       a[j] = a[k];
       a[k] = temp;
   } else {
       if (idx == 1) {
           temp = a[i];
           a[i] = a[j];
           a[j] = a[k];
           a[k] = temp;
       } else {
           if (idx == 2) {
               temp = a[i];
               a[i] = a[j];
               a[j] = a[k];
               a[k] = temp;
           }
       }
   }
}
 
int main() {
   int a[10], i;
   hipMalloc(&a, 10 * sizeof(int));
   hipMemcpy(a, a, 10 * sizeof(int), hipMemcpyHostToDevice);
   for (i = 0; i < 10; i++) {
       scanf("%d", &a[i]);
   }
   hipFree(a);
   return 0;
}

#include <hip/hip_runtime.h>
#include <stdio.h>
 
__global__ void odd_even_sort(int *a, int N)
{
   int idx = threadIdx.x;
   int offset = idx * 2;
   int i = offset;
   int j = offset + 1;
   int k = offset + 2;
   int temp;
   if (idx == 0) {
       temp = a[i];
       a[i] = a[j];
       a[j] = a[k];
       a[k] = temp;
   } else {
       if (idx == 1) {
           temp = a[i];
           a[i] = a[j];
           a[j] = a[k];
           a[k] = temp;
       } else {
           if (idx == 2) {
               temp = a[i];
               a[i] = a[j];
               a[j] = a[k];
               a[k] = temp;
           }
       }
   }
}
 
int main() {
   int a[10], i;
   hipMalloc(&a, 10 * sizeof(int));
   hipMemcpy(a, a, 10 * sizeof(int), hipMemcpyHostToDevice);
   for (i = 0; i < 10; i++) {
       scanf("%d", &a[i]);
   }
   hipFree(a);
   return 0;
}

#include <hip/hip_runtime.h>
#include <stdio.h>
 
__global__ void odd_even_sort(int *a, int N)
{
   int idx = threadIdx.x;
   int offset = idx * 2;
   int i = offset;
   int j = offset + 1;
   int k = offset + 2;
   int temp;
   if (idx == 0) {
       temp = a[i];
       a[i] = a[j];
       a[j] = a[k];
       a[k] = temp;
   } else {
       if (idx == 1) {
           temp = a[i];
           a[i] = a[j];
           a[j] = a[k];
           a[k] = temp;
       } else {
           if (idx == 2) {
               temp = a[i];
               a[i]