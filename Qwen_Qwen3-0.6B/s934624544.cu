#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void odd_even_transpose(int *a, int *b, int N){
  int i = threadIdx.x;
  int j = threadIdx.y;
  int k = threadIdx.z;
  int idx = i + j * N + k * N;
  if (idx < N) {
    a[idx] = b[idx];
  } else {
    a[idx] = a[idx - N];
  }
}

int main() {
  int N = 9;
  int *a = (int*)malloc(N * sizeof(int));
  int *b = (int*)malloc(N * sizeof(int));
  hipMalloc(&a, N * sizeof(int));
  hipMalloc(&b, N * sizeof(int));
  hipMemcpy(a, &i, N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(b, &j, N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(a, &sum, N * sizeof(int), hipMemcpyHostToDevice);
  hipFree(a);
  hipFree(b);
  return 0;
}

#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void odd_even_transpose(int *a, int *b, int N){
  int i = threadIdx.x;
  int j = threadIdx.y;
  int k = threadIdx.z;
  int idx = i + j * N + k * N;
  if (idx < N) {
    a[idx] = b[idx];
  } else {
    a[idx] = a[idx - N];
  }
}

int main() {
  int N = 9;
  int *a = (int*)malloc(N * sizeof(int));
  int *b = (int*)malloc(N * sizeof(int));
  hipMalloc(&a, N * sizeof(int));
  hipMalloc(&b, N * sizeof(int));
  hipMemcpy(a, &i, N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(b, &j, N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(a, &sum, N * sizeof(int), hipMemcpyHostToDevice);
  hipFree(a);
  hipFree(b);
  return 0;
}

#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void odd_even_transpose(int *a, int *b, int N){
  int i = threadIdx.x;
  int j = threadIdx.y;
  int k = threadIdx.z;
  int idx = i + j * N + k * N;
  if (idx < N) {
    a[idx] = b[idx];
  } else {
    a[idx] = a[idx - N];
  }
}

int main() {
  int N = 9;
  int *a = (int*)malloc(N * sizeof(int));
  int *b = (int*)malloc(N * sizeof(int));
  hipMalloc(&a, N * sizeof(int));
  hipMalloc(&b, N * sizeof(int));
  hipMemcpy(a, &i, N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(b, &j, N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(a, &sum, N * sizeof(int), hipMemcpyHostToDevice);
  hipFree(a);
  hipFree(b);
  return 0;
}

#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void odd_even_transpose(int *a, int *b, int N){
  int i = threadIdx.x;
  int j = threadIdx.y;
  int k = threadIdx.z;
  int idx = i + j * N + k * N;
  if (idx < N) {
    a[idx] = b[idx];
  } else {
    a[idx] = a[idx - N];
  }
}

int main() {
  int N = 9;
  int *a = (int*)malloc(N * sizeof(int));
  int *b = (int*)malloc(N * sizeof(int));
  hipMalloc(&a, N * sizeof(int));
  hipMalloc(&b, N * sizeof(int));
  hipMemcpy(a, &i, N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(b, &j, N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(a, &sum, N * sizeof(int), hipMemcpyHostToDevice);
  hipFree(a);
  hipFree(b);
  return 0;
}

#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void odd_even_transpose(int *a, int *b, int N){
  int i = threadIdx.x;
  int j = threadIdx.y;
  int k = threadIdx.z;
  int idx = i + j * N + k * N;
  if (idx < N) {
    a[idx] = b[idx];
  } else {
    a[idx] = a[idx - N];
  }
}

int main() {
  int N = 9;
  int *a = (int*)malloc(N * sizeof(int));
  int *b = (int*)